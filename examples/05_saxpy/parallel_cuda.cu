// Copyright (c) 2016 Thomas Heller
//
// Distributed under the Boost Software License, Version 1.0. (See accompanying
// file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <hpx/hpx_init.hpp>
#include <hpx/util/high_resolution_timer.hpp>
#include <hpx/include/compute.hpp>
#include <hpx/include/parallel_transform.hpp>

#include <boost/program_options.hpp>

#include <algorithm>
#include <vector>
#include <iostream>

int hpx_main(boost::program_options::variables_map& vm)
{
    std::size_t N = vm["N"].as<std::size_t>();
    std::size_t steps = vm["steps"].as<std::size_t>();

    typedef hpx::compute::cuda::default_allocator allocator_type;
    typedef hpx::compute::cuda::default_executor executor_type;

    hpx::compute::cuda::target device;

    allocator_type alloc(device);

    hpx::compute::vector<double, allocator_type> a(N, 0.0, alloc);
    hpx::compute::vector<double, allocator_type> b(N, 1.0, alloc);
    hpx::compute::vector<double, allocator_type> c(N, 2.0, alloc);
    double x = 3.0;

    executor_type exec(device);

    hpx::util::high_resolution_timer t;
    for (std::size_t t = 0; t < steps; ++t)
    {
        using hpx::parallel::execution::par;
        hpx::parallel::transform(par.on(exec),
            b.begin(), b.end(), c.begin(), a.begin(),
            [x] HPX_DEVICE (double bb, double cc)
            {
                return bb * x + cc;
            }
        );
    }
    double elapsed = t.elapsed();

    double bandwidth = ((steps * N * sizeof(double)) / elapsed) / 1e6;
    std::cout << "Bandwidth: " << bandwidth << " MB/s\n";

    return hpx::finalize();
}

int main(int argc, char* argv[])
{
    using namespace boost::program_options;

    options_description desc_commandline;
    desc_commandline.add_options()
        ("N", value<std::size_t>()->default_value(1024),
         "Number of Elements")
        ("steps", value<std::size_t>()->default_value(100),
         "Number of steps to apply the stencil")
    ;

    return hpx::init(desc_commandline, argc, argv);
}
